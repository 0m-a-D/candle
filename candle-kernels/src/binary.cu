#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"
#include<stdint.h>

#if __CUDA_ARCH__ >= 800
BINARY_OP(__hip_bfloat16, badd_bf16, x + y)
BINARY_OP(__hip_bfloat16, bdiv_bf16, x / y)
BINARY_OP(__hip_bfloat16, bmul_bf16, x * y)
BINARY_OP(__hip_bfloat16, bsub_bf16, x - y)
#endif

#if __CUDA_ARCH__ >= 530
BINARY_OP(__half, badd_f16, x + y)
BINARY_OP(__half, bdiv_f16, x / y)
BINARY_OP(__half, bmul_f16, x * y)
BINARY_OP(__half, bsub_f16, x - y)
#endif

BINARY_OP(float, badd_f32, x + y)
BINARY_OP(double, badd_f64, x + y);
BINARY_OP(uint8_t, badd_u8, x + y);
BINARY_OP(uint32_t, badd_u32, x + y);
BINARY_OP(float, bdiv_f32, x / y)
BINARY_OP(double, bdiv_f64, x / y);
BINARY_OP(uint8_t, bdiv_u8, x / y);
BINARY_OP(uint32_t, bdiv_u32, x / y);
BINARY_OP(float, bmul_f32, x * y)
BINARY_OP(double, bmul_f64, x * y);
BINARY_OP(uint8_t, bmul_u8, x * y);
BINARY_OP(uint32_t, bmul_u32, x * y);
BINARY_OP(float, bsub_f32, x - y)
BINARY_OP(double, bsub_f64, x - y);
BINARY_OP(uint8_t, bsub_u8, x - y);
BINARY_OP(uint32_t, bsub_u32, x - y);
