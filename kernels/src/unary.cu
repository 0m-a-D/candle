#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

#define UNARY_OP(TYPENAME, FN_NAME, FUNC) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
        TYPENAME x = inp ? inp[i] : out[i]; \
        out[i] = FUNC; \
    } \
} \

#if __CUDA_ARCH__ >= 530
UNARY_OP(__half, uneg_f16, -x)
UNARY_OP(__half, usqr_f16, x*x)
UNARY_OP(__half, usqrt_f16, sqrtg(x))
#endif

UNARY_OP(float, uneg_f32, -x)
UNARY_OP(float, uneg_f64, -x)
UNARY_OP(float, usqr_f32, x*x)
UNARY_OP(float, usqr_f64, x*x)
UNARY_OP(float, usqrt_f32, sqrtg(x))
UNARY_OP(float, usqrt_f64, sqrtg(x))
