#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

#define UNARY_OP(TYPENAME, FN_NAME, FUNC) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
        unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
        TYPENAME x = inp ? inp[strided_i] : out[i]; \
        out[i] = FUNC; \
    } \
} \

template<typename T>
__device__ T gelu_fwd(T x) {
    T x_sq = x * x;
    T x_cube = x_sq * x;
    T alpha = x + static_cast<T>(0.044715) * x_cube;
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + tanhg(static_cast<T>(M_2_SQRTPI * M_SQRT1_2) * alpha));
}


#if __CUDA_ARCH__ >= 530
UNARY_OP(__half, ucopy_f16, x)
UNARY_OP(__half, uneg_f16, -x)
UNARY_OP(__half, usqr_f16, x*x)
UNARY_OP(__half, usqrt_f16, sqrtg(x))
UNARY_OP(__half, gelu_f16, gelu_fwd(x))
#endif

UNARY_OP(float, ucopy_f32, x)
UNARY_OP(double, ucopy_f64, x)
UNARY_OP(float, uneg_f32, -x)
UNARY_OP(double, uneg_f64, -x)
UNARY_OP(float, usqr_f32, x*x)
UNARY_OP(double, usqr_f64, x*x)
UNARY_OP(float, usqrt_f32, sqrtg(x))
UNARY_OP(double, usqrt_f64, sqrtg(x))
UNARY_OP(float, gelu_f32, gelu_fwd(x))
UNARY_OP(double, gelu_f64, gelu_fwd(x))
