#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

#define CAST_OP(SRC_TYPENAME, DST_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const SRC_TYPENAME *inp, \
    DST_TYPENAME *out \
) { \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            out[i] = inp[i]; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            out[i] = inp[strided_i]; \
        } \
    } \
} \

#if __CUDA_ARCH__ >= 530
CAST_OP(__half, __half, cast_f16_f16)
CAST_OP(__half, float, cast_f16_f32)
CAST_OP(float, __half, cast_f32_f16)
#endif

CAST_OP(float, float, cast_f32_f32)
CAST_OP(float, double, cast_f32_f64)
CAST_OP(double, float, cast_f64_f32)
