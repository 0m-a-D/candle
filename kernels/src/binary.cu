#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

#if __CUDA_ARCH__ >= 530
BINARY_OP(__half, badd_f16, x + y)
BINARY_OP(__half, bdiv_f16, x / y)
BINARY_OP(__half, bmul_f16, x * y)
BINARY_OP(__half, bsub_f16, x - y)
#endif

BINARY_OP(float, badd_f32, x + y)
BINARY_OP(double, badd_fwd_f64, x + y);
BINARY_OP(float, bdiv_f32, x / y)
BINARY_OP(double, bdiv_f64, x / y);
BINARY_OP(float, bmul_f32, x * y)
BINARY_OP(double, bmul_f64, x * y);
BINARY_OP(float, bsub_f32, x - y)
BINARY_OP(double, bsub_f64, x - y);
